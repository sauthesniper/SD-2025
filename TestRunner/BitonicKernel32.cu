// bitonic_kernel.cu

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>

using namespace std;
using namespace std::chrono;

__global__ void BitonicSort(int* v, int size, int stage, int pass) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	size /= 2;

	if(i >= size) return;

	int group = i / pass;
	int dir = i / (stage / 2) % 2;

	int li = i + group * pass;
	int ri = li + pass;

	int aux = li;
	// Math for if(dir == 0) swap(li, ri)
	aux = ri * dir + li * (1 - dir);
	ri = li * dir + ri * (1 - dir);
	li = aux;

	int minval = min(v[li], v[ri]);
	int maxval = max(v[li], v[ri]);
	v[li] = minval;
	v[ri] = maxval;
}

__global__ void BitonicSortSharedMemory(int* v, int size, int stage, int pass) {
	// Thread group memory; Avoids reading from global memory
	extern __shared__ int sharedMem[];

	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + tid;

	size /= 2;

	if(i >= size) return;

	// Load data into shared memory
	sharedMem[tid] = v[i];
	__syncthreads();

	int group = i / pass;
	int dir = i / (stage / 2) % 2;

	int li = i + group * pass;
	int ri = li + pass;

	bool swap = (dir == 0) ? (sharedMem[li] > sharedMem[ri]) : (sharedMem[li] < sharedMem[ri]);
	if (swap) {
		int aux = sharedMem[li];
		sharedMem[li] = sharedMem[ri];
		sharedMem[ri] = aux;
	}

	__syncthreads();

	v[i] = sharedMem[tid];
}

void LaunchCudaSortKernel(int* cpuData, int size) {
	int* gpuData;

	// Warm up CUDA drivers
	hipMalloc(0, 0);

	auto start = high_resolution_clock::now();
	// Allocate on the gpu
	hipMalloc((void**)&gpuData, size * sizeof(int));
	auto end = high_resolution_clock::now();
	auto duration = duration_cast<milliseconds>(end - start);
	cout << "GPU memory allocation took " << duration.count() << "ms\n";

	start = high_resolution_clock::now();
	hipMemcpy(gpuData, cpuData, size * sizeof(int), hipMemcpyHostToDevice);
	end = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(end - start);
	cout << "Memory transfer to GPU took " << duration.count() << "ms\n";

	int blockSize = 256;
	int numBlocks = (size + blockSize - 1) / blockSize;


	start = high_resolution_clock::now();
	for(int s = 2; s <= size; s *= 2) {
		for(int p = s / 2; p > 0; p /= 2) {
			BitonicSort<<<numBlocks, blockSize>>>(gpuData, size, s, p);

			// Wait for the gpu to finish processing
			hipDeviceSynchronize();
		}
	}
	end = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(end - start);
	cout << "Sorting on GPU took " << duration.count() << "ms\n";

	start = high_resolution_clock::now();
	// Copy data from gpu
	hipMemcpy(cpuData, gpuData, size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(gpuData);
	end = high_resolution_clock::now();
	duration = duration_cast<milliseconds>(end - start);
	cout << "Memory transfer to CPU took " << duration.count() << "ms\n";
}


int* ReadNumbersFromFile(const char* filePath, int* size){
	FILE* file = fopen(filePath, "r");
	if(file == NULL){
		perror("Errror opening file\n");
		fclose(file);
		return NULL;
	}

	if(fscanf(file, "%d", size)!=1){
		perror("Error reading size from file\n");
		fclose(file);
		return NULL;
	}

	int* arr = (int*)malloc((*size) * sizeof(int));
	if(arr == NULL){
		perror("Memory allocation failed\n");
		fclose(file);
		return NULL;
	}

	for(int i = 0; i < *size; i++){
		if(fscanf(file, "%d", &arr[i]) != 1){
			perror("Error reading numbers from file\n");
			free(arr);
			fclose(file);
			return NULL;
		}
	}

	fclose(file);
	return arr;
}
int main(int argc, char* argv[]){
	if(argc < 2){
		fprintf(stderr, "Usage: %s <filename0> [filename1...]\n", argv[0]);
		return 0;
	}
	for(int i = 1; i < argc; i++) {
		const char* filePath = argv[i];
		int size;
		int* arr = ReadNumbersFromFile(filePath, &size);

		if(arr == NULL) continue;

		printf("Running test file %s...\n", filePath);

		LaunchCudaSortKernel(arr, size);

		/*for(int j = 0; j < size; j++){
			printf("%i ", arr[j]);
		}
		printf("\n\n");*/

		free(arr);
	}
	return 0;
}

